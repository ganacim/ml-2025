#include "hip/hip_runtime.h"
#include "timer.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif
#include "kernel.h"



std::vector<int> cuda_histogram(const std::vector<float>& v, int num_bins, util::timer_pool& timers) {
    if (v.empty()) {
        std::cerr << "Error: Input vector is empty." << std::endl;
        return {};
    }

    float *d_v = nullptr;
    int *d_hist = nullptr;
    int vector_size = static_cast<int>(v.size());

    float min_val = *std::min_element(v.begin(), v.end());
    float max_val = *std::max_element(v.begin(), v.end());

    std::cout << "CUDA Min: " << min_val << ", Max: " << max_val << std::endl;

    hipMalloc(&d_v, vector_size * sizeof(float));
    hipMalloc(&d_hist, num_bins * sizeof(int));

    hipMemset(d_hist, 0, num_bins * sizeof(int));

    hipMemcpy(d_v, v.data(), vector_size * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (vector_size + BLOCK_SIZE - 1) / BLOCK_SIZE;

    auto& cuda_timer = timers.gpu_add("CUDA Histogram Computation", v.size());
    cuda_timer.do_start();

    histogram_kernel<<<blocks, BLOCK_SIZE>>>(d_v, d_hist, num_bins, vector_size, min_val, max_val);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel Error: " << hipGetErrorString(err) << std::endl;
        return {};
    }

    hipDeviceSynchronize();

    cuda_timer.do_stop();

    std::vector<int> hist_host(num_bins);
    hipMemcpy(hist_host.data(), d_hist, num_bins * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_v);
    hipFree(d_hist);

    return hist_host;
}
