#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include <stdio.h>

using namespace std;

// Define a device function, which 
// can be called from a kernel and executes on the GPU
__device__ int device_function(){
    printf("Hello CUDA World!\n");
    return 1;
}

// Define a kernel function, which is the entry point
// for execution on the GPU
__global__ void kernel() {
    device_function();
}

// Define a wrapper function, which launches the kernel
void kernel_wrapper() {
    // Launch kernel with <<<block, thread>>> syntax
    kernel<<<1,32>>>();
}

vector<float> cuda_multiplication(const std::vector<float>& m1,
                                    const std::vector<float>& m2,
                                    unsigned int m1_rows,
                                    unsigned int m1_cols,
                                    unsigned int m2_cols)
{
}