#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "timer.h"
#include <stdio.h>
#include <iostream>
#include <exception>
#include <cmath>
#include <vector>

using namespace std;

// Define a kernel function, which is the entry point
// for execution on the GPU


#define BLOCK_SIZE 32

__global__ void matrix_conv_wkernel(const float *m_in, 
                            const float *kernel, 
                            float *result, 
                            unsigned int m_rows, 
                            unsigned int m_cols,
                            unsigned int kernel_size)
{
    // Get the row and column of the current element
    unsigned int ti = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + ti;
    unsigned int tj = threadIdx.y;
    unsigned int j = blockIdx.y*blockDim.y + tj;

    if (i < m_cols-kernel_size+1 && j < m_rows-kernel_size+1){
        //int k_center = kernel_size / 2;
        float value = 0.0f;

        for (unsigned int x=0; x < kernel_size; x++) {
            for (unsigned int y=0; y < kernel_size; y++) { 
                int mrow = j + y;
                int mcol = i + x;

                if (mrow < m_rows && mcol < m_cols) {
                    value += m_in[mrow * m_cols + mcol] * kernel[x * kernel_size + y];
                }
            }
            
        }
    if (j < m_rows-kernel_size+1 && i < m_cols-kernel_size+1){
        result[j * (m_cols-kernel_size+1) + i] = value;
    }
    }
}

vector<float> cuda_convolution_template(const std::vector<float>& m,
                                const std::vector<float>& k,
                                unsigned int rows,
                                unsigned int cols,
                                unsigned int kernel_size)
{
    string name = "CUDA Conv2d";
    auto& timer = util::timers.gpu_add(name);

    // Allocate memory on the host
    int out_cols = cols - kernel_size  + 1;
    int out_rows = rows - kernel_size + 1;
    vector<float> result(out_rows * out_cols);

    // Allocate memory on the device
    float *d_m, *d_kern, *d_result;
    hipMalloc(&d_m, rows * cols * sizeof(float));
    hipMalloc(&d_kern, kernel_size * kernel_size * sizeof(float));
    hipMalloc(&d_result, out_rows * out_cols * sizeof(float));
    // Copy data from host to device
    hipMemcpy(d_m, m.data(), rows * cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kern, k.data(), kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice);
    // // sync cuda device
    // hipDeviceSynchronize();
    // Define grid and block size
    int grid_size_y = (out_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int grid_size_x = (out_cols + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 grid(grid_size_x, grid_size_y);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    matrix_conv_wkernel<<<grid, block>>>(d_m, d_kern, d_result, rows, cols, kernel_size);
    
    // Launch kernel
    // // sync cuda device
    // hipDeviceSynchronize();
    // Copy data from device to host
    hipMemcpy(result.data(), d_result, out_rows * out_cols * sizeof(float), hipMemcpyDeviceToHost);
    // Free memory on the device
    hipFree(d_m);
    hipFree(d_kern);
    hipFree(d_result);
    timer.stop();
    return result;
}