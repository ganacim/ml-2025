#include "hip/hip_runtime.h"
#include <vector>
#include "kernel.h"

#include <stdio.h>
#include <iostream>

#include <cuda/std/complex>
#include <random>

typedef std::mt19937 RNG;  // Mersenne Twister with a popular choice of parameters

using namespace cuda::std;

const int BLOCK_SIZE = 16;
const int MAX_ITER = 2000;
// Define a kernel function, which is the entry point
// for execution on the GPU
__global__ void mandelbrot_kernel(int *d_res, const int WIDTH, const int HEIGHT, const float scale, const float cx, const float cy) {
    unsigned int ti = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + ti;
    unsigned int tj = threadIdx.y;
    unsigned int j = blockIdx.y*blockDim.y + tj;
    
    if (i >= WIDTH || j >= HEIGHT){
        return;
    }

    const double x = ((double) i/WIDTH - 0.5) * scale + cx;
    const double y = ((double) j/HEIGHT - 0.5) * scale + cy;

    complex<double> c(x, y), z(0, 0);

    int k = 0;
    while(abs(z) < 2 && k < MAX_ITER){
        z = z*z + c;
        k++;
    }

    if (k < 4){
        d_res[j * WIDTH + i] = 3;
    } else if(k == MAX_ITER){
        d_res[j * WIDTH + i] = 10;
    } else{
        d_res[j * WIDTH + i] = k % 7;
    }

    return;
}

// Define a wrapper function, which launches the kernel
void kernel_wrapper(int* result, const int WIDTH, const int HEIGHT, const float scale, const float cx, const float cy) {
    // Launch kernel with <<<block, thread>>> syntax
    int *d_result;
    hipMalloc(&d_result, WIDTH * HEIGHT * sizeof(int));

    dim3 grid(ceil((float)WIDTH/BLOCK_SIZE), ceil((float)HEIGHT/BLOCK_SIZE), 1);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    
    mandelbrot_kernel<<<grid, block>>>(d_result, WIDTH, HEIGHT, scale, cx, cy);

    hipMemcpy(result, d_result, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);
}
