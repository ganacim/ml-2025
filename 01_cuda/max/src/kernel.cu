#include "hip/hip_runtime.h"
#include "kernel.h"
#include <vector>

#include <iostream>
#include <stdio.h>

#include <ctime>
#include <random>

typedef std::mt19937
    RNG; // Mersenne Twister with a popular choice of parameters
using namespace std;

#define BLOCK_SIZE 32

__global__ void max_kernel(float *d_v, float *d_max, int excess_size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int ti = threadIdx.x;
  __shared__ float sdata[BLOCK_SIZE];

  sdata[ti] = d_v[i];
  __syncthreads();

  float max_val = sdata[0];
  if (blockIdx.x == gridDim.x - 1) {
    if (ti == 0) {
      for (int k = 1; k < BLOCK_SIZE - excess_size; k++) {
        max_val = max(max_val, sdata[k]);
      }
    }
  } else {
    if (ti == 0) {
      for (int k = 1; k < BLOCK_SIZE; k++) {
        max_val = max(max_val, sdata[k]);
      }
    }
  }
  d_max[blockIdx.x] = max_val;
}

vector<float> create_random_vector(unsigned int cols) {
  // Create a normal distribution with mean 0 and standard deviation 1
  uint32_t seed = (uint32_t)time(0);
  RNG rng(seed);
  normal_distribution<float> normal(0.0, 1.0);
  // Create a matrix of size matrix_size x matrix_size with random values
  vector<float> matrix(cols);
  for (int j = 0; j < cols; j++) {
    matrix[j] = normal(rng);
  }
  return matrix;
}

void kernel_wrapper() {
  vector<float> v = create_random_vector(1e9);
  // vector<float> v = {-2.0, -3.0, -1.0, -4.0};

  // cout << "Original v: ";
  // for (auto i : v) {
  //   cout << i << " ";
  //
  float *d_v;
  hipMalloc(&d_v, v.size() * sizeof(float));
  hipMemcpy(d_v, v.data(), v.size() * sizeof(float), hipMemcpyHostToDevice);

  int length_max_vector;

  // NOTE: For a, b integers: (a + (b - 1)) / b = ceil(a/b)
  length_max_vector = (v.size() + (BLOCK_SIZE - 1)) / BLOCK_SIZE;
  int excess_size;
  excess_size = length_max_vector * BLOCK_SIZE - v.size();

  float *d_max;
  hipMalloc(&d_max, length_max_vector * sizeof(float));

  while (length_max_vector >= 1) {

    dim3 grid(length_max_vector);
    dim3 block(BLOCK_SIZE);

    max_kernel<<<grid, block>>>(d_v, d_max, excess_size);

    float *aux;
    aux = d_max;
    d_max = d_v;
    d_v = aux;
    if (length_max_vector == 1) {
      length_max_vector = 0;
    }
    int new_length_max;
    new_length_max = (length_max_vector + (BLOCK_SIZE - 1)) / BLOCK_SIZE;
    excess_size = new_length_max * BLOCK_SIZE - length_max_vector;
    length_max_vector = new_length_max;
  }

  float final_max;
  hipMemcpy(&final_max, d_v, sizeof(float), hipMemcpyDeviceToHost);
  cout << "Final max value: " << final_max << endl;

  // // Real max value
  // float max_val = v[0];
  // for (int i = 1; i < v.size(); i++) {
  //   max_val = max(max_val, v[i]);
  // }
  // cout << "Real max value: " << max_val << endl;

  hipFree(d_v);
}
