#include "hip/hip_runtime.h"
#include "kernel.h"
#include <vector>

#include <iostream>
#include <stdio.h>

#include <ctime>
#include <random>

typedef std::mt19937
    RNG; // Mersenne Twister with a popular choice of parameters
using namespace std;

#define BLOCK_SIZE 3

__global__ void max_kernel(float *d_v, float *d_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int ti = threadIdx.x;
  __shared__ float sdata[BLOCK_SIZE];

  sdata[ti] = d_v[i];
  __syncthreads();

  float max_val = sdata[0];
  if (ti == 0) {
    for (int k = 1; k < BLOCK_SIZE; k++) {
      max_val = max(max_val, sdata[k]);
    }
  }
  d_max[blockIdx.x] = max_val;
}

vector<float> create_random_vector(unsigned int cols) {
  // Create a normal distribution with mean 0 and standard deviation 1
  uint32_t seed = (uint32_t)time(0);
  RNG rng(seed);
  normal_distribution<float> normal(0.0, 1.0);
  // Create a matrix of size matrix_size x matrix_size with random values
  vector<float> matrix(cols);
  for (int j = 0; j < cols; j++) {
    matrix[j] = normal(rng);
  }
  return matrix;
}

// FIXME: Not working when BLOCK_SIZE doesn't divide the size of the vector

void kernel_wrapper() {
  vector<float> v = create_random_vector(4);

  cout << "Original v: ";
  for (auto i : v) {
    cout << i << " ";
  }
  cout << endl;

  float *d_v;
  hipMalloc(&d_v, v.size() * sizeof(float));
  hipMemcpy(d_v, v.data(), v.size() * sizeof(float), hipMemcpyHostToDevice);

  int length_max_vector;

  // NOTE: For a, b integers: (a + (b - 1)) / b = ceil(a/b)
  length_max_vector = (v.size() + (BLOCK_SIZE - 1)) / BLOCK_SIZE;

  while (length_max_vector > 1) {
    float *d_max;
    hipMalloc(&d_max, length_max_vector * sizeof(float));

    dim3 grid(length_max_vector);
    dim3 block(BLOCK_SIZE);

    max_kernel<<<grid, block>>>(d_v, d_max);

    d_v = d_max;
    length_max_vector = (length_max_vector + (BLOCK_SIZE - 1)) / BLOCK_SIZE;
  }
  float final_max;
  hipMemcpy(&final_max, d_v, sizeof(float), hipMemcpyDeviceToHost);
  cout << "Final max value: " << final_max << endl;

  // Real max value
  float max_val = v[0];
  for (int i = 1; i < v.size(); i++) {
    max_val = max(max_val, v[i]);
  }
  cout << "Real max value: " << max_val << endl;

  hipFree(d_v);
}
